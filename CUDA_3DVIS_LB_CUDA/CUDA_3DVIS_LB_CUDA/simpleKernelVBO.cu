#include "hip/hip_runtime.h"
#pragma once

#ifndef __BASIC_INTEROP_H__
 #define __BASIC_INTEROP_H__

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

typedef struct
{
	float4 pos;
	float4 color;
	float4 dir_speed;
} Vertex;

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(Vertex* v, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
	int index = y * width + x;
	if( v[index].pos.x < -0.99f )
	{
		v[index].dir_speed.x = 1.0f;
	}
	else if( v[index].pos.x > 0.99f )
	{
		v[index].dir_speed.x = -1.0f;
	}
	v[index].pos.x += (0.001f*v[index].dir_speed.x*v[index].dir_speed.z);

	if( v[index].pos.y < -0.99f )
	{
		v[index].dir_speed.y = 1.0f;
	}
	else if( v[index].pos.y > 0.99f )
	{
		v[index].dir_speed.y = -1.0f;
	}
	v[index].pos.y += (0.001f*v[index].dir_speed.y*v[index].dir_speed.z);

	//Para verlo en 3D:
	v[index].pos.z = sin(time*v[index].dir_speed.z);
}

extern "C" void runCuda(hipGraphicsResource** resource, Vertex* devPtr, int dim, float dt)
{
	//Getting an actual address in device memory that can be passed to our kernel. 
	//We achieve this by instructing the CUDA runtime to map the
	//shared resource and then by requesting a pointer to the mapped resource.
    checkCudaErrors( hipGraphicsMapResources( 1, resource, NULL ) );
    // devPtr is our device memory
    size_t  size;
    checkCudaErrors( hipGraphicsResourceGetMappedPointer( (void**)&devPtr, &size, *resource) );

	//launchKernel (devPtr, DIM, dt);
    dim3    numBlocks(dim/16,dim/16);
    dim3    numThreads(16,16);
    kernel<<<numBlocks,numThreads>>>( devPtr, dim, dim, dt );

	//unmapping our shared resource. This call is important to make prior to performing rendering tasks because it
	//provides synchronization between the CUDA and graphics portions of the application. Specifically, 
	//it implies that all CUDA operations performed prior to the call
	//to hipGraphicsUnmapResources() will complete before ensuing graphics
	//calls begin.
	checkCudaErrors( hipGraphicsUnmapResources( 1, resource, NULL ) );
}

extern "C" void unregRes(hipGraphicsResource** res)
{
	checkCudaErrors( hipGraphicsUnmapResources( 1, res, NULL ) );
}

extern "C" void chooseDev(int ARGC, const char **ARGV)
{
	gpuGLDeviceInit(ARGC, ARGV);
}

extern "C" void regBuffer(hipGraphicsResource** res, unsigned int& vbo)
{
	// setting up graphics interoperability by notifying the CUDA runtime 
	//that we intend to share the OpenGL buffer named vbo with CUDA.
	checkCudaErrors( hipGraphicsGLRegisterBuffer( res, vbo, cudaGraphicsMapFlagsWriteDiscard ) );
}

#endif
